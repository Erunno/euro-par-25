#include "hip/hip_runtime.h"
#include "gol.cuh"
#include <hip/hip_runtime.h>
#include <iostream>
#include "tile-macro.hpp"

#define WORD_TYPE uint64_t

__device__ WORD_TYPE load_word(const WORD_TYPE* input, int x_word, int y_word, int x_dim, int y_dim) {
    if (x_word < 0 || x_word >= x_dim || y_word < 0 || y_word >= y_dim) {
        return 0;
    }
    return input[y_word * x_dim + x_word];
}

__global__ void gol_kernel(const WORD_TYPE* input, WORD_TYPE* output, int dim) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int x_dim = dim / 8;
    int y_dim = dim / 8;
    
    if (x >= x_dim || y >= y_dim) return;

    WORD_TYPE tl, tc, tr;
    WORD_TYPE cl, cc, cr;
    WORD_TYPE bl, bc, br;

    tl = load_word(input, x - 1, y - 1, x_dim, y_dim);
    tc = load_word(input, x,     y - 1, x_dim, y_dim);
    tr = load_word(input, x + 1, y - 1, x_dim, y_dim);
    cl = load_word(input, x - 1, y,     x_dim, y_dim);
    cc = load_word(input, x,     y,     x_dim, y_dim);
    cr = load_word(input, x + 1, y,     x_dim, y_dim);
    bl = load_word(input, x - 1, y + 1, x_dim, y_dim);
    bc = load_word(input, x,     y + 1, x_dim, y_dim);
    br = load_word(input, x + 1, y + 1, x_dim, y_dim);

    WORD_TYPE result = GOL_COMPUTE_TILED(
        tl, tc, tr,
        cl, cc, cr,
        bl, bc, br);

    output[y * x_dim + x] = result;
}

void run_game_of_life(const std::uint64_t* input, std::uint64_t* output, int grid_dimensions) {
    int x_dim = grid_dimensions / 8;
    int y_dim = grid_dimensions / 8;

    dim3 blockSize(16, 16);
    dim3 gridSize((x_dim + blockSize.x - 1) / blockSize.x, (y_dim + blockSize.y - 1) / blockSize.y);

    gol_kernel<<<gridSize, blockSize>>>(input, output, grid_dimensions);
}

void run_game_of_life(const std::uint32_t* input, std::uint32_t* output, int grid_dimensions) {}
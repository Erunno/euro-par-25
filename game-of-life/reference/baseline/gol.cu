#include "hip/hip_runtime.h"
#include "gol.cuh"
#include <hip/hip_runtime.h>
#include <iostream>

#define INDEX(x, y, dim) ((y) * (dim) + (x))

__global__ void gol_kernel(const uint32_t* input, uint32_t* output, int dim) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= dim || y >= dim) return;

    int count = 0;
    
    int x_start = (x == 0) ? 0 : -1;
    int x_end = (x == dim - 1) ? 0 : 1;
    int y_start = (y == 0) ? 0 : -1;
    int y_end = (y == dim - 1) ? 0 : 1;
    
    for (int dx = x_start; dx <= x_end; dx++) {
        for (int dy = y_start; dy <= y_end; dy++) {
            
            if (dx == 0 && dy == 0) continue;

            int nx = x + dx;
            int ny = y + dy;

            count += input[INDEX(nx, ny, dim)];
        }
    }

    int current = input[INDEX(x, y, dim)];
    int new_state = (count == 3 || (current && count == 2)) ? 1 : 0;

    output[INDEX(x, y, dim)] = new_state;
}

void run_game_of_life(const std::uint32_t* input, std::uint32_t* output, int grid_dimensions) {
    uint32_t *d_input, *d_output;
    size_t size = grid_dimensions * grid_dimensions * sizeof(uint32_t);

    hipMalloc(&d_input, size);
    hipMalloc(&d_output, size);
    hipMemcpy(d_input, input, size, hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((grid_dimensions) / 16, (grid_dimensions) / 16);

    gol_kernel<<<gridSize, blockSize>>>(d_input, d_output, grid_dimensions);

    hipMemcpy(output, d_output, size, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}
